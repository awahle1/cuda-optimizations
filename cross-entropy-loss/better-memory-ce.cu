#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <float.h>

#define C 512
#define N 1048576

float ce_loss_sequential(float * L, int* Y){
    float total_log_likelyhood = 0;
    for (int i=0; i<N; ++i){
        float exp_logit = expf(L[i*C + Y[i]]);

        float softmax_denominator = 0;
        for (int z = 0; z<C; ++z){
            softmax_denominator += expf(L[i*C + z]);
        }
        float log_likelyhood = logf(exp_logit/softmax_denominator);
        total_log_likelyhood += log_likelyhood;
    }

    return -total_log_likelyhood / N;
}

float randomFloat(float min, float max) {
  return min + (float)rand() / RAND_MAX * (max - min);
}

void fill_L(float * L){
    for (int i=0; i<N; ++i){
        for (int j=0; j<C; ++j){
            L[i*C + j] = randomFloat(-1, 1);
        }
    }
}

void fill_Y(int* Y){
    for (int i=0; i<N; ++i){
        Y[i] = 0;
    }
}

__global__
void naiveCrossEntropy(float * L, int* Y, float * loss){
    //One Block Per Row
    int row = blockIdx.x;

    if (row>=N){
        return;
    }

    __shared__ int class_ind;
    __shared__ float logit_val;
    __shared__ float sum_exp; 
    if (threadIdx.x == 0) {
        class_ind = Y[row];
        sum_exp = 0.0f;
    }

    // __syncthreads();

    float thread_sum = 0; 
    for (int phase=0; phase<C/blockDim.x; ++phase){
        if (threadIdx.x + phase*blockDim.x < C){
            float val = expf(L[row*C + threadIdx.x + phase*blockDim.x]);
            thread_sum += val;
            if (threadIdx.x + phase*blockDim.x == class_ind){
                logit_val = val;
            }
        }
        
    }

    // atomicAdd(&sum_exp, thread_sum);

    // __syncthreads();

    // if (threadIdx.x == 0){
    //     float log_likelihood = logf(logit_val/sum_exp);
    //     atomicAdd(loss, -log_likelihood/N);
    // }


}

int main() {
    float * h_L = (float*)malloc(C*N*sizeof(float));fill_L(h_L);
    float * d_L;hipMalloc(&d_L, C*N*sizeof(float));
    hipMemcpy(d_L, h_L, C*N*sizeof(float), hipMemcpyHostToDevice);

    int * h_Y = (int*)malloc(N*sizeof(int));fill_Y(h_Y);
    int * d_Y;hipMalloc(&d_Y, N*sizeof(int));
    hipMemcpy(d_Y, d_Y, N*sizeof(int), hipMemcpyHostToDevice);

    float* d_loss; 
    hipMalloc(&d_loss, sizeof(float));
    hipMemset(d_loss, 0, sizeof(float));

    int blockSize = 32;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float ms = 0.0f;

    // Kernel Timing
    hipEventRecord(start);
    naiveCrossEntropy<<<N, blockSize>>>(d_L, d_Y, d_loss);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    printf("Kernel Execution Time ms: %f \n", ms);
    // END

    float zero = 0;
    float* h_loss = &zero; 
    hipMemcpy(h_loss, d_loss, sizeof(float), hipMemcpyDeviceToHost);

    float correct_loss = ce_loss_sequential(h_L, h_Y);
    printf("Correct Loss: %f, Kernel Loss: %f\n", correct_loss, *h_loss);

    hipFree(d_loss);
    hipFree(d_L);
    hipFree(d_Y);
    free(h_L);
    free(h_Y);
}